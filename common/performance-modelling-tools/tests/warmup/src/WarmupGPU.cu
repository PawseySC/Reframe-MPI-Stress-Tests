#include "hip/hip_runtime.h"
#include <WarmupGPU.h>
#include <iostream>
#include <cmath>

#if defined(USEHIP) || defined(USECUDA)
/// standard scalar a * vector x plus vector y 
__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}


/// just a vector add to new vector
__global__
void vector_add(float *out, float *a, float *b, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if (id < n) out[id] = a[id] + b[id];
}


/// silly kernel that should be very quick
__global__
void silly_kernel(float *a)
{
    for (int i = 0; i < 2; i++) {
        a[i] + 2*a[i];
    }
}
#endif

void launch_warmup_kernel(int itype, int i, int j, unsigned long long N) 
{
    std::string kernel_type, s;
    float t1;
    float *a, *b;
    if (itype == GPU_ONLY_KERNEL_LAUNCH)
    {
        kernel_type = "KernelLaunchOnly";
        s = kernel_type + " on device " + std::to_string(i) + " round " + std::to_string(j); 
        auto mytimer = NewTimer();
#ifdef _OPENMP 
        #pragma omp target
        {
            #pragma omp parallel for
            for (int i = 0; i < 2; i++) {a[i] + 2*a[i];}
        }
        
        // N = 200000000;
        // std::cout<<"memory "<<N*sizeof(float)/1024./1024./1024.<<std::endl;
        // a = new float[N];
        // // #pragma omp target map(tofrom:a[:N])
        // #pragma omp target map(alloc:a[:N])
        // {
        //     #pragma omp parallel for
        //     for (int i = 0; i < N; i++) {
        //         a[i] = 1.0;
        //         a[i] = a[i] + 2*a[i];
        //         a[i] = exp(-sqrt(a[i]));
        //     }
        // }
        std::cout<<s<<" ";
        LogTimeTaken(mytimer);
#elif defined(_OPENACC)
        #pragma acc parallel loop
            for (int i = 0; i < 2; i++) {a[i] + 2*a[i];}
        std::cout<<s<<" ";
        LogTimeTaken(mytimer);
#elif defined(USEHIP) || defined(USECUDA)
        silly_kernel<<<1,1>>>(a);
        LogGPUElapsedTime(s, mytimer);
#endif
    }
    if (itype == GPU_ONLY_MEM_ALLOCATE) 
    {
        kernel_type = "MemAllocOnly";
        s = kernel_type + " on device " + std::to_string(i) + " round " + std::to_string(j); 
        auto mytimer = NewTimer();
#ifdef _OPENMP
        // auto a_d = omp_target_alloc(N*sizeof(float), i);
        // omp_target_free(a_d, i);
        #pragma omp target data map(alloc:a[:N])
        {
        }
        std::cout<<s<<" ";
        LogTimeTaken(mytimer);
#elif _OPENACC
        #pragma acc kernels create(a[:N])
        {
        }
        std::cout<<s<<" ";
        LogTimeTaken(mytimer);
#elif defined(USEHIP) || defined(USECUDA)
        gpuMalloc(&a, N*sizeof(float)); 
        gpuFree(a);
        LogGPUElapsedTime(s, mytimer);
#endif
    }
    else if (itype == GPU_ONLY_MEM_TH2D) 
    {
        kernel_type = "tH2D";
        s = kernel_type + " on device " + std::to_string(i) + " round " + std::to_string(j); 
        a = new float[N];
#ifdef _OPENMP
        auto mytimer = NewTimer();
        #pragma omp target data map(to:a[:N])
        {
            
        }
        std::cout<<s<<" ";
        LogTimeTaken(mytimer);
#elif defined(_OPENACC)
        auto mytimer = NewTimer();
        #pragma acc kernels copyin(a[:N])
        {
            
        }
        std::cout<<s<<" ";
        LogTimeTaken(mytimer);
#elif defined(USEHIP) || defined(USECUDA)
        gpuMalloc(&b, N*sizeof(float)); 
        auto mytimer = NewTimer();
        gpuMemcpy(b, a, N*sizeof(float), gpuMemcpyHostToDevice);
        LogGPUElapsedTime(s, mytimer);
        gpuFree(b);
#endif
        delete[] a;
    }
    // transfer from device to host
    else if (itype == GPU_ONLY_MEM_TD2H) 
    {
        kernel_type = "tD2H";
        s = kernel_type + " on device " + std::to_string(i) + " round " + std::to_string(j); 
        a = new float[N];
#ifdef _OPENMP
        auto mytimer = NewTimer();
        #pragma omp target data map(from:a[:N])
        {

        }
        std::cout<<s<<" ";
        LogTimeTaken(mytimer);
#elif defined(_OPENACC)
        auto mytimer = NewTimer();
        #pragma acc kernels copyout(a[:N])
        {
            
        }
        std::cout<<s<<" ";
        LogTimeTaken(mytimer);
#elif defined(USEHIP) || defined(USECUDA)
        gpuMalloc(&b, N*sizeof(float)); 
        auto mytimer = NewTimer();
        gpuMemcpy(a, b, N*sizeof(float), gpuMemcpyDeviceToHost);
        LogGPUElapsedTime(s, mytimer);
        gpuFree(b);
#endif
        delete[] a;
    }
}

inline int GetNumDevices()
{
    int deviceCount = 0;
#ifdef _OPENMP 
    deviceCount = omp_get_num_devices();
#elif defined(_OPENACC)
    auto dtype = acc_get_device_type();
    deviceCount = acc_get_num_devices(dtype);
#elif defined(USEHIP) || defined(USECUDA)
    gpuGetDeviceCount(&deviceCount);
#endif
    return deviceCount;
}

inline void SetDevice(int i)
{
#ifdef _OPENMP 
    omp_set_default_device(i);
#elif defined(_OPENACC)
    auto dtype = acc_get_device_type();
    acc_set_device_num(i,dtype);
#elif defined(USEHIP) || defined(USECUDA)
    gpuSetDevice(i);
#endif
}

void warmup_kernel_over_kernels(int rounds, 
    std::vector<int> kernel_order,
    unsigned long long N
)
{
    int deviceCount = GetNumDevices();
    std::cout<<__func__<<" running "<<std::endl;
    auto timeWarmup = NewTimer();
    for (auto i=0;i<deviceCount;i++) 
    {
        // set the device 
        SetDevice(i);
        for (auto j=0;j<rounds;j++) 
        {
            for (auto itype:kernel_order) 
            {
                launch_warmup_kernel(itype, i, j, N);
            }
        }
    }
    LogTimeTaken(timeWarmup);
}

void warmup_kernel_over_rounds(int rounds, int sleeptime, unsigned long long N)
{
    int deviceCount = GetNumDevices();
    std::cout<<__func__<<" running "<<std::endl;
    auto timeWarmup = NewTimer();
    for (auto itype=0;itype<GPU_ONLY_NUM_LAUNCH_TYPES;itype++) 
    {
        for (auto i=0;i<deviceCount;i++) 
        {
            // set the device 
            SetDevice(i);
            for (auto j=0;j<rounds;j++) 
            {
                launch_warmup_kernel(itype, i, j, N);
                if (sleeptime > 0) sleep(sleeptime);
            }
        }
    }
    LogTimeTaken(timeWarmup);
}

void run_on_devices(Logger &logger, int Niter)
{
    int deviceCount = GetNumDevices();
    for (auto i=0;i<deviceCount;i++) 
    {
        SetDevice(i);
        // now check the kernel launches
        std::vector<double> times;
        std::map<std::string, std::vector<double>> device_times;
        std::vector<double> x;
#ifdef _OPENMP 
        device_times.insert({"omp_target",x});
#elif defined(_OPENACC)
        device_times.insert({"acc_target",x});
#elif defined(USEHIP) || defined(USECUDA) 
        device_times.insert({"allocation",x});
        device_times.insert({"tH2D",x});
        device_times.insert({"tD2H",x});
        device_times.insert({"free",x});
        device_times.insert({"kernel",x});
#endif
        for (auto j=0;j<Niter;j++) 
        {
            auto t = NewTimer();
            auto timings = run_kernel(j);
            times.push_back(t.get());
            for (auto &t:timings) 
            {
                device_times[t.first].push_back(t.second);
            }
        }
        std::cout<<"================================="<<std::endl;
        std::cout<<" DEVICE "<<i<<std::endl;
        logger.ReportTimes("run_kernel", times);
        std::cout<<"---------------------------------"<<std::endl;
        std::cout<<"On device times within run_kernel"<<std::endl;
        for (auto &t:device_times) logger.ReportTimes(t.first,t.second);
        std::cout<<"---------------------------------"<<std::endl;
       
    }
}

std::map<std::string, double> run_kernel(int offset)
{
#define gettelapsed(t1)  telapsed = GetTimeTakenOnDevice(t1,__func__, std::to_string(__LINE__));

    std::map<std::string, double> timings;
    float telapsed;
    auto N = 1024*1024;
    float *x, *y, *d_x, *d_y, *out, *d_out;
    x = new float[N];
    y = new float[N];
    out = new float[N];
    for (int i = 0; i < N; i++) 
    {
        x[i] = 1.0*offset;
        y[i] = 2.0*offset;
    }
#ifdef _OPENMP 
    auto tall = NewTimer();
    // auto talloc = NewTimer();
    // auto th2d = NewTimer();
    // auto tk = NewTimer();
    // auto td2h = NewTimer();
    // auto tfree = NewTime();
    #pragma omp target data map(to:x[:N],y[:N]) map(from:out[:N]) 
    {
        #pragma omp target
        #pragma omp parallel for
        for (int i=0;i<N;i++) out[i] = x[i] + y[i];
    }
    // timings.insert({std::string("allocation"), telapsed});
    // timings.insert({std::string("tH2D"), telapsed});
    // timings.insert({std::string("kernel"), telapsed});
    // timings.insert({std::string("tD2H"), telapsed});
    // timings.insert({std::string("free"), telapsed});
    // std::cout<<out[2]<<std::endl;
    // LogTimeTaken(tall);
    telapsed = GetTimeTaken(tall,__func__, std::to_string(__LINE__));
    timings.insert({std::string("omp_target"), telapsed});
#elif defined(_OPENACC)
    auto tall = NewTimer();
    #pragma acc parallel loop copyin(x[:N],y[:N]) copyout(out[:N])
    for (int i=0;i<N;i++) out[i] = x[i] + y[i];
    telapsed = GetTimeTaken(tall,__func__, std::to_string(__LINE__));
    timings.insert({std::string("acc_target"), telapsed});
#elif defined(USEHIP) || defined(USECUDA)
    auto talloc = NewTimer();
    gpuMalloc(&d_x, N*sizeof(float)); 
    gpuMalloc(&d_y, N*sizeof(float));
    gpuMalloc(&d_out, N*sizeof(float));
    gettelapsed(talloc);
    timings.insert({std::string("allocation"), telapsed});
    auto th2d = NewTimer();
    gpuMemcpy(d_x, x, N*sizeof(float), gpuMemcpyHostToDevice);
    gpuMemcpy(d_y, y, N*sizeof(float), gpuMemcpyHostToDevice);
    gettelapsed(th2d);
    timings.insert({std::string("tH2D"), telapsed});
    ///\todo need to update kernel launch to use something other 
    /// than <<<1,1>>>
    auto tk = NewTimer();
    int blockSize, gridSize;
    // Number of threads in each thread block
    blockSize = 1024;
    // Number of thread blocks in grid
    gridSize = static_cast<int>(ceil(static_cast<float>(N)/blockSize));
    // Execute the kernel
    vector_add<<<dim3(gridSize),dim3(blockSize)>>>(d_out, d_x, d_y, N);
    gettelapsed(tk);
    timings.insert({std::string("kernel"), telapsed});
    auto td2h = NewTimer();
    gpuMemcpy(out, d_out, N*sizeof(float), gpuMemcpyDeviceToHost);
    gettelapsed(td2h);
    timings.insert({std::string("tD2H"), telapsed});
    auto tfree = NewTimer();
    gpuFree(d_x);
    gpuFree(d_y);
    gpuFree(d_out);
    gettelapsed(tfree);
    timings.insert({std::string("free"), telapsed});
#endif
    delete[] x;
    delete[] y;
    delete[] out;

    return timings;
}
